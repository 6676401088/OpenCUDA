#include "hip/hip_runtime.h"
// SimpleBrightnessGradient.cu
// 实现简单亮度渐变算法

#include "SimpleBrightnessGradient.h"                                          
#include <math.h> 

#include <iostream> 
using namespace std;

#include "ErrorCode.h" 

// 宏：DEF_BLOCK_X 和 DEF_BLOCK_Y
// 定义了默认的线程块的尺寸。
#define DEF_BLOCK_X  16 
#define DEF_BLOCK_Y  16 

// Kernel 函数： SimpleBrightnessGradient （简单亮度渐变） 
static __global__ void    // Kernel 函数无返回值
_simpleBrightnessGradientKer( 
        ImageCuda inimg,  // 输入图像
        ImageCuda outimg  // 输出图像
);

// Kernel 函数：SimpleBrightnessGradient（简单亮度渐变）
static __global__ void _simpleBrightnessGradientKer(ImageCuda inimg, 
                                                    ImageCuda outimg) 
{ 
    // c 和 r 分别表示线程处理的像素点的坐标的 x 和 y 分量 
    //（其中，c 表示 column， r 表示 row）。由于采用并行度缩减策略 ，
    // 令一个线程处理 4 个输出像素，这四个像素位于统一列的相邻 4 行上，
    // 因此，对于r 需要进行乘 4 的计算
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = (blockIdx.y * blockDim.y + threadIdx.y) * 4;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，
    //一方面节省计算资源，一方面防止由于段错误导致的程序崩溃。
    if (c >= inimg.imgMeta.width || r >= inimg.imgMeta.height) 
        return;

    // 计算输入坐标点对应的图像数据数组下标。
    int inidx = r * inimg.pitchBytes + c;

    // 计算输出坐标点对应的图像数据数组下标。
    int outidx = r * outimg.pitchBytes + c;

    // 读取输入坐标点对应的像素值
    unsigned char inpixel;
    inpixel = inimg.imgMeta.imgData[inidx];

    unsigned char intemp;
    const unsigned char temp = sqrtf(inimg.imgMeta.width * 
                               inimg.imgMeta.width + 
                               inimg.imgMeta.height * inimg.imgMeta.height);
    intemp = (1 - (1 / temp) * sqrtf(c * c + r * r)) * 255 + 
             (1 / temp) * sqrtf(c * c + r * r) * inpixel;

    //输出图像像素
    outimg.imgMeta.imgData[outidx] = intemp;

    // 处理剩下的 3 个点
    for(int i = 1; i < 4; i++) { 
        // 这三个像素点，每个像素点都在前一个的下一行，而 x 分量保持不变。因
        // 此，需要检查这个像素点是否越界。检查只针对 y 分量即可，x 分量在各
        // 点之间没有变化，故不用检查
        if (r + i > inimg.imgMeta.height) 
            return;

        // 获取当前列的下一行的位置指针
        inidx += inimg.pitchBytes;
        outidx += outimg.pitchBytes;

        intemp = (1 - (1 / temp) * sqrtf(c * c + r * r)) * 255 +  
                 (1 / temp) * sqrtf(c * c + r * r) * inpixel;

        //输出图像像素
        outimg.imgMeta.imgData[outidx] = intemp;
    }
}

// Host 成员方法: SimpleBrightnessGradient（简单亮度渐变）
__host__ int SimpleBrightnessGradient::simpleBrightnessGradient(Image *inimg, 
                                                                Image *outimg) 
{ 
    // 检查输入图像是否为 NULL，如果为 NULL 直接报错返回。
    if (inimg == NULL || outimg == NULL) 
        return NULL_POINTER;

    // 这一段代码进行图像的预处理工作。图像的预处理主要完成在 Device 内存上为
    // 输入和输出图像准备内存空间，以便盛放数据。
    int errcode;  // 局部变量，错误码

    // 将输入图像拷贝到 Device 内存中。
    errcode = ImageBasicOp::copyToCurrentDevice(inimg);
    if (errcode != NO_ERROR) 
        return errcode;

    // 将输出图像拷贝入 Device 内存。
    errcode = ImageBasicOp::copyToCurrentDevice(outimg);
    if (errcode != NO_ERROR) { 
        // 计算 roi 子图的宽和高
        int roiwidth = inimg->roiX2 - inimg->roiX1; 
        int roiheight = inimg->roiY2 - inimg->roiY1;

        // 如果输出图像无数据（故上面的拷贝函数会失败），则会创建一个和输入图
        // 像的 ROI 子图像尺寸相同的图像。
        errcode = ImageBasicOp::makeAtCurrentDevice(outimg, roiwidth, 
                                                    roiheight); 
        // 如果创建图像也操作失败，则说明操作彻底失败，报错退出。
        if (errcode != NO_ERROR) 
            return errcode;
    }

    // 提取输入图像的 ROI 子图像。
    ImageCuda insubimgCud;
    errcode = ImageBasicOp::roiSubImage(inimg, &insubimgCud);
    if (errcode != NO_ERROR) 
        return errcode;
    
    // 提取输出图像的 ROI 子图像。
    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) 
        return errcode;

    // 计算调用 Kernel 函数的线程块的尺寸和线程块的数量。
    dim3 blocksize, gridsize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / 
                  blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y * 4 - 1) / 
                 (blocksize.y * 4);

    // 调用核函数
    _simpleBrightnessGradientKer<<<gridsize, blocksize>>>(insubimgCud, 
                                                         outsubimgCud);
    if (hipGetLastError() != hipSuccess) 
        return CUDA_ERROR;
    
    // 处理完毕，退出。
    return NO_ERROR;
} 

