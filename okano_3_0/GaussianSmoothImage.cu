#include "hip/hip_runtime.h"
﻿// GaussianSmoothImage.cu
// 图像高斯平滑操作，包括普通高斯平滑和带mask的高斯平滑

#include "GaussianSmoothImage.h"
#include "ErrorCode.h"

// 宏定义，定义了五个高斯平滑尺度对应的权重总和
#define GAUSS_THREE  16
#define GAUSS_FIVE   256
#define GAUSS_SEVEN  4096
#define GAUSS_NINE   65536 
#define GAUSS_ELEVEN 1048576 

// 宏：DEF_BLOCK_X 和 DEF_BLOCK_Y
// 定义了默认的线程块的尺寸。
#define DEF_BLOCK_X  32
#define DEF_BLOCK_Y   8

// 下列五个核函数为普通高斯平滑核函数

// 1.平滑窗口大小为3*3的高斯平滑函数
static __global__ void
gauss3SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标
    int smWidth,                // 平滑窗口宽度
    int smHeight                // 平滑窗口高度
); 

// 2.平滑窗口大小为5*5的高斯平滑函数                       
static __global__ void
gauss5SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标
    int smWidth,                // 平滑窗口宽度
    int smHeight                // 平滑窗口高度
); 
// 3.平滑窗口大小为7*7的高斯平滑函数                                   
static __global__ void
gauss7SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标
    int smWidth,                // 平滑窗口宽度
    int smHeight                // 平滑窗口高度
); 

// 4.平滑窗口大小为9*9的高斯平滑函数 
static __global__ void
gauss9SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标
    int smWidth,                // 平滑窗口宽度
    int smHeight                // 平滑窗口高度
); 

// 5.平滑窗口大小为11*11的高斯平滑函数                       
static __global__ void
gauss11SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标
    int smWidth,                // 平滑窗口宽度
    int smHeight                // 平滑窗口高度
); 

//  下列五个核函数为带mask的高斯平滑函数

// 1.平滑窗口大小为3*3，带mask的高斯平滑函数    
static __global__ void
gauss3SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标 
    int smWidth,                // 平滑窗口宽度
    int smHeight,               // 平滑窗口高度
    ImageCuda maskImageGPU,     // mask图像
    unsigned char mask          // mask值
);

// 2.平滑窗口大小为5*5，带mask的高斯平滑函数    
static __global__ void
gauss5SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标 
    int smWidth,                // 平滑窗口宽度
    int smHeight,               // 平滑窗口高度
    ImageCuda maskImageGPU,     // mask图像
    unsigned char mask          // mask值
);

// 3.平滑窗口大小为7*7，带mask的高斯平滑函数    
static __global__ void
gauss7SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标 
    int smWidth,                // 平滑窗口宽度
    int smHeight,               // 平滑窗口高度
    ImageCuda maskImageGPU,     // mask图像
    unsigned char mask          // mask值
);

// 4.平滑窗口大小为9*9，带mask的高斯平滑函数    
static __global__ void
gauss9SmoothImage(
    ImageCuda const origiImageGPU ,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标 
    int smWidth,                // 平滑窗口宽度
    int smHeight,               // 平滑窗口高度
    ImageCuda maskImageGPU,     // mask图像
    unsigned char mask          // mask值
);

// 5.平滑窗口大小为11*11，带mask的高斯平滑函数    
static __global__ void
gauss11SmoothImage(
    ImageCuda origiImageGPU,    // 原始图像
    ImageCuda gaussSmImageGPU,  // 平滑后图像
    int smLocatX,               // 平滑起始横坐标
    int smLocatY,               // 平滑起始纵坐标 
    int smWidth,                // 平滑窗口宽度
    int smHeight,               // 平滑窗口高度 
    ImageCuda maskImageGPU,     // mask图像
    unsigned char mask          // mask值
);



 // 平滑窗口大小为7*7的高斯平滑函数实现    
static __global__ void gauss7SmoothImage(ImageCuda origiImageGPU, 
                                          ImageCuda gaussSmImageGPU, 
                                          int smLocatX, int smLocatY,
                                          int smWidth, int smHeight)
{
      // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
    if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 高斯平滑系数数组
     int GF[7] = {1, 6, 15, 20, 15, 6, 1};
     
    // 高斯卷积累加和
    int c = 0;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
    for(int i = 0; i < 7;i++) {
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
        for(int j = 0; j < 7;j++)
            c += GF[i] * GF[j] *
                 origiImageGPU.imgMeta.imgData[(y + i - 3) * w + (x + j - 3)];
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / GAUSS_SEVEN + 0.5f; 
}

 // 平滑窗口大小为5*5的高斯平滑函数实现
static __global__ void gauss5SmoothImage(ImageCuda origiImageGPU, 
                                          ImageCuda gaussSmImageGPU, 
                                          int smLocatX, int smLocatY,
                                          int smWidth, int smHeight)
{
    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
    if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 高斯平滑系数数组
    int GF[5] = {1, 4, 6, 4, 1};

    // 高斯卷积累加和
    int c = 0;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
    for(int i = 0; i < 5;i++) {
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
        for(int j = 0; j < 5;j++)
            c += GF[i] * GF[j] *
                 origiImageGPU.imgMeta.imgData[(y + i - 2) * w + (x + j - 2)];
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / GAUSS_FIVE  + 0.5f; 
}

// 平滑窗口大小为9*9的高斯平滑函数实现
static __global__ void gauss9SmoothImage(ImageCuda origiImageGPU, 
                                          ImageCuda gaussSmImageGPU, 
                                          int smLocatX, int smLocatY,
                                          int smWidth, int smHeight)
{
    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
    if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 高斯平滑系数数组
    const int GF[9] = {1, 8, 28, 56, 70, 56, 28, 8, 1};

    // 高斯卷积累加和
    int c = 0;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
    for(int i = 0; i < 9;i++) {
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
         for(int j = 0; j < 9; j++)
            c += GF[i] * GF[j] *
                 origiImageGPU.imgMeta.imgData[(y + i - 4) * w + (x + j - 4)];
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / GAUSS_NINE  + 0.5f; 
}

// 平滑窗口大小为11*11的高斯平滑函数实现
static __global__ void gauss11SmoothImage(ImageCuda origiImageGPU, 
                                          ImageCuda gaussSmImageGPU, 
                                          int smLocatX, int smLocatY,
                                          int smWidth, int smHeight)
{
    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
    if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 高斯平滑系数数组
    int GF[11] = {1, 10, 45, 120, 210, 252, 210, 120, 45, 10, 1};

    // 高斯卷积累加和
    int c = 0;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
    for(int i = 0; i < 11;i++) {
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll

         for(int j = 0; j < 11;j++)
            c += GF[i] * GF[j] *
                 origiImageGPU.imgMeta.imgData[(y + i - 5) * w + (x + j - 5)];
    }
    
     // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / GAUSS_ELEVEN  + 0.5f; 
}

 // 平滑窗口大小为3*3的高斯平滑函数实现
static __global__ void gauss3SmoothImage(ImageCuda origiImageGPU, 
                                          ImageCuda gaussSmImageGPU, 
                                          int smLocatX, int smLocatY,
                                          int smWidth, int smHeight)
{

    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
    if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 高斯平滑系数数组
    int GF[3] = {1, 2, 1};

    // 高斯卷积累加和
    int c = 0;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
    for(int i = 0; i < 3;i++) {
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll

         for(int j = 0; j < 3; j++)
            c += GF[i] * GF[j] *
                 origiImageGPU.imgMeta.imgData[(y + i - 1) * w + (x + j - 1)];
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / GAUSS_THREE  + 0.5f ; 
}


// 平滑窗口大小为7*7的，带mask高斯平滑函数实现
static __global__ void gauss7SmoothImage(ImageCuda origiImageGPU, 
                                         ImageCuda gaussSmImageGPU, 
                                         int smLocatX, int smLocatY, 
                                         int smWidth, int smHeight,
                                         ImageCuda maskImageGPU,
                                         unsigned char mask)
{
    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 如果mask图像像素值不等于mask则不处理
    if (maskImageGPU.imgMeta.imgData[y * w + x]  != mask) 
        return ;
        
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
     if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 获取mask图像数据
    unsigned char * maskImg = maskImageGPU.imgMeta.imgData;
    
    // 高斯平滑系数数组
    int gf[7] = {1, 6, 15, 20, 15, 6, 1};
    
    // 高斯卷积累加和 
    int c = 0;
    
    // 参加计算的像素点权重总和wsum,当前权重wgh
    int wsum = 0, wgh;
    
    // 图像像素索引
    int mIdx;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll 
 
    for(int i = 0; i < 7; i++){
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
          for(int j = 0; j < 7; j++) {  
            // 获取图像像素索引
            mIdx=(y + i - 3) * w + (x + j - 3);
            
            // 只处理mask图像像素值等于mask值的像素点
            if (maskImg[mIdx] == mask) {
            
                // 计算当前像素点的权重
                wgh = gf[i] * gf[j];
                
                // 当前像素点的权重累加到总权重中
                wsum += wgh ;
                
                // 计算像素值加权累加和
                c += wgh * origiImageGPU.imgMeta.imgData[mIdx]; 
            }
        }
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / wsum  + 0.5f; 
}

 // 平滑窗口大小为5*5的，带mask高斯平滑函数实现
static __global__ void gauss5SmoothImage(ImageCuda origiImageGPU, 
                                         ImageCuda gaussSmImageGPU, 
                                         int smLocatX, int smLocatY, 
                                         int smWidth, int smHeight,
                                         ImageCuda maskImageGPU,
                                         unsigned char mask)
{
    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 如果mask图像像素值不等于mask则不处理
    if (maskImageGPU.imgMeta.imgData[y * w + x]  != mask) 
        return ;
        
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
     if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 获取mask图像数据
    unsigned char * maskImg = maskImageGPU.imgMeta.imgData;
    
    // 高斯平滑系数数组
    int gf[5] = {1, 4, 6, 4, 1};
    
    // 高斯卷积累加和 
    int c = 0;
    
    // 参加计算的像素点权重总和wsum,当前权重wgh
    int wsum = 0, wgh;
    
    // 图像像素索引
    int mIdx;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll 
 
    for(int i = 0; i < 5; i++){
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
          for(int j = 0; j < 5; j++) {  
            // 获取图像像素索引
            mIdx=(y + i - 2) * w + (x + j - 2);
            
            // 只处理mask图像像素值等于mask值的像素点
            if (maskImg[mIdx] == mask) {
            
                // 计算当前像素点的权重
                wgh = gf[i] * gf[j];
                
                // 当前像素点的权重累加到总权重中
                wsum += wgh ;
                
                // 计算像素值加权累加和
                c += wgh * origiImageGPU.imgMeta.imgData[mIdx]; 
            }
        }
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / wsum  + 0.5f; 
}

 // 平滑窗口大小为9*9的，带mask高斯平滑函数实现
static __global__ void gauss9SmoothImage(ImageCuda origiImageGPU, 
                                         ImageCuda gaussSmImageGPU, 
                                         int smLocatX, int smLocatY, 
                                         int smWidth, int smHeight,
                                         ImageCuda maskImageGPU,
                                         unsigned char mask)
{
    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 如果mask图像像素值不等于mask则不处理
    if (maskImageGPU.imgMeta.imgData[y * w + x]  != mask) 
        return ;
        
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
    if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 获取mask图像数据
    unsigned char * maskImg = maskImageGPU.imgMeta.imgData;
    
    // 高斯平滑系数数组
    int gf[9] =  {1, 8, 28, 56, 70, 56, 28, 8, 1};
    
    // 高斯卷积累加和 
    int c = 0;
    
    // 参加计算的像素点权重总和wsum,当前权重wgh
    int wsum = 0, wgh;
    
    // 图像像素索引
    int mIdx;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll 
 
    for(int i = 0; i < 9; i++){
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
          for(int j = 0; j < 9; j++) {  
            // 获取图像像素索引
            mIdx=(y + i - 4) * w + (x + j - 4);
            
            // 只处理mask图像像素值等于mask值的像素点
            if (maskImg[mIdx] == mask) {
            
                // 计算当前像素点的权重
                wgh = gf[i] * gf[j];
                
                // 当前像素点的权重累加到总权重中
                wsum += wgh ;
                
                // 计算像素值加权累加和
                c += wgh * origiImageGPU.imgMeta.imgData[mIdx]; 
            }
        }
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / wsum  + 0.5f; 
}

 // 平滑窗口大小为11*11的，带mask高斯平滑函数实现
static __global__ void gauss11SmoothImage(ImageCuda origiImageGPU,              
                                          ImageCuda gaussSmImageGPU, 
                                          int smLocatX, int smLocatY, 
                                          int smWidth, int smHeight,
                                          ImageCuda maskImageGPU,
                                          unsigned char mask)
{
    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 如果mask图像像素值不等于mask则不处理
    if (maskImageGPU.imgMeta.imgData[y * w + x]  != mask) 
        return ;
        
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
     if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 获取mask图像数据
    unsigned char * maskImg = maskImageGPU.imgMeta.imgData;
    
    // 高斯平滑系数数组
    int gf[11] = {1, 10, 45, 120, 210, 252, 210, 120, 45, 10, 1};
    
    // 高斯卷积累加和 
    int c = 0;
    
    // 参加计算的像素点权重总和wsum,当前权重wgh
    int wsum = 0, wgh;
    
    // 图像像素索引
    int mIdx;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll 
 
    for(int i = 0; i < 11; i++) {
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
         for(int j = 0; j < 11; j++) {  
            // 获取图像像素索引
            mIdx=(y + i - 5) * w + (x + j - 5);
            
            // 只处理mask图像像素值等于mask值的像素点
            if (maskImg[mIdx] == mask) {
            
                // 计算当前像素点的权重
                wgh = gf[i] * gf[j];
                
                // 当前像素点的权重累加到总权重中
                wsum += wgh ;
                
                // 计算像素值加权累加和
                c += wgh * origiImageGPU.imgMeta.imgData[mIdx]; 
            }
        }
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / wsum  + 0.5f; 
}

 // 平滑窗口大小为3*3的，带mask高斯平滑函数实现
static __global__ void gauss3SmoothImage(ImageCuda origiImageGPU, 
                                         ImageCuda gaussSmImageGPU, 
                                         int smLocatX, int smLocatY, 
                                         int smWidth, int smHeight,
                                         ImageCuda maskImageGPU,
                                         unsigned char mask)
{
    // 获取pixel在原图像中的位置 
    int w = origiImageGPU.pitchBytes;
    int x = blockIdx.x * blockDim.x + threadIdx.x + smLocatX;
    int y = blockIdx.y * blockDim.y + threadIdx.y + smLocatY;
    
    // 如果mask图像像素值不等于mask则不处理
    if (maskImageGPU.imgMeta.imgData[y * w + x]  != mask) 
        return ;
        
    // 检查像素点是否越界，如果越界，则不进行处理，一方面节省计算资
    // 源，一方面防止由于段错误导致的程序崩溃。
     if(x >= smLocatX + smWidth || y >= smLocatY + smHeight)
        return ;

    // 获取mask图像数据
    unsigned char * maskImg = maskImageGPU.imgMeta.imgData;
    
    // 高斯平滑系数数组
    int gf[3] = {1, 2, 1};
    
    // 高斯卷积累加和 
    int c = 0;
    
    // 参加计算的像素点权重总和wsum,当前权重wgh
    int wsum = 0, wgh;
    
    // 图像像素索引
    int mIdx;
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll 
 
    for(int i = 0; i < 3; i++) {
    
    // 编译预处理，在编译阶段将循环展开，节约循环跳转时间
    #pragma unroll
    
         for(int j = 0; j < 3; j++) {  
            // 获取图像像素索引
            mIdx=(y + i - 1) * w + (x + j - 1);
            
            // 只处理mask图像像素值等于mask值的像素点
            if (maskImg[mIdx] == mask) {
            
                // 计算当前像素点的权重
                wgh = gf[i] * gf[j];
                
                // 当前像素点的权重累加到总权重中
                wsum += wgh ;
                
                // 计算像素值加权累加和
                c += wgh * origiImageGPU.imgMeta.imgData[mIdx]; 
            }
        }
    }
    
    // 计算平滑后像素值，结果四舍五入
    gaussSmImageGPU.imgMeta.imgData[y * w + x] = 1.0 * c / wsum  + 0.5f; 

}

// 普通高斯平滑函数
__host__ int GaussSmoothImage::gaussSmoothImage(Image* origiImage, int smWidth, 
                                                int smHeight, int smLocatX, 
                                                int smLocatY, int smWindowSize, 
                                                Image* gaussSmImage)
{
    // 局部变量，错误码。
    int errcode;  

    // 输入输出图像指针不能为空
    if (origiImage == NULL || gaussSmImage == NULL)
        return NULL_POINTER;

    // 获取图像尺寸信息
    int  imgWidth = origiImage->width;
    int  imgHeight = origiImage->height;

    // 图像小于平滑范围
    if (imgWidth < smWidth || imgHeight < smHeight) 
        return -11; 
        
    // 平滑范围小于最大平滑窗口大小
    if (smWidth < 11 || smHeight < 11) 
        return -12;
        
    // 输入的平滑窗口大小不在处理范围之内
    if (smWindowSize < 3 || smWindowSize > 11) 
        return -13;

    // 平滑计算所涉及data位置或范围不能超出原始图像的物理范围，
    // 故应根据smWindowSize作适当调整。
    int  marginOff = (smWindowSize + 1) >> 1;
    int  leftMargin = smLocatX - marginOff;
    int  rightMargin = imgWidth - smLocatX - smWidth - marginOff;

    int  topMargin = smLocatY - marginOff;
    int  bottomMargin = imgHeight - smLocatY - smHeight - marginOff;
 
    // 平滑时将发生左侧出界
    if (leftMargin < 0) {
        smLocatX -= leftMargin;
        smWidth += leftMargin;
    }

    // 平滑时将发生右侧出界
    if (rightMargin < 0) {
        smWidth += rightMargin;
    }
    
    // 平滑宽度小于1
    if (smWidth < 1)
        return -14; 

    // 平滑时将发生上方出界
    if (topMargin < 0) {
        smLocatY -= topMargin;
        smHeight += topMargin;
    }

    // 平滑时将发生下方出界
    if (bottomMargin < 0) {
        smHeight += bottomMargin;
    }

    // 平滑高度小于1
    if (smHeight < 1) 
        return -15;  
        
    // 将输入图像拷贝到 Device 内存中。
    errcode = ImageBasicOp::copyToCurrentDevice(origiImage);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    // 将输出图像拷贝到 Device 内存中。
    errcode = ImageBasicOp::copyToCurrentDevice(gaussSmImage);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    // 提取输入图像。
    ImageCuda origiImageGPU;
    errcode = ImageBasicOp::roiSubImage(origiImage, &origiImageGPU);
    if (errcode != NO_ERROR) {
        return errcode;
    }
    
    // 提取输出图像。
    ImageCuda gaussSmImageGPU;
    errcode = ImageBasicOp::roiSubImage(gaussSmImage, &gaussSmImageGPU);
    if (errcode != NO_ERROR) {
        return errcode;
    }
    
    // 计算调用 Kernel 函数的线程块的尺寸和线程块的数量。
    dim3 gridSize,blockSize;
    
    blockSize.x = DEF_BLOCK_X;
    blockSize.y = DEF_BLOCK_Y;
    gridSize.x = (smWidth + blockSize.x - 1) / blockSize.x;
    gridSize.y = (smHeight + blockSize.y - 1) / blockSize.y;
    
    // 根据平滑窗口大小选择对应的核函数
    // 按照委托方要求，顺序为7、5、9、11、3
    switch (smWindowSize) { 
    
    case 7: 
    // 启动平滑窗口大小为7的核函数
        gauss7SmoothImage<<<gridSize, blockSize>>>(origiImageGPU, 
                                                   gaussSmImageGPU,
                                                   smLocatX, smLocatY,
                                                   smWidth, smHeight);
        // 核函数出错                                         
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }    
         
        break;
 
    case 5:
        // 启动平滑窗口大小为5的核函数
        gauss5SmoothImage<<<gridSize, blockSize>>>(origiImageGPU, 
                                                   gaussSmImageGPU,
                                                   smLocatX, smLocatY,
                                                   smWidth, smHeight);
        // 核函数出错                                         
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }
            
        break;
     
    case 9:
           
        // 启动平滑窗口大小为9的核函数
        gauss9SmoothImage<<<gridSize, blockSize>>>(origiImageGPU, 
                                                   gaussSmImageGPU,
                                                   smLocatX, smLocatY,
                                                   smWidth, smHeight);               
        // 核函数出错                                         
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }
      
        break;
         
    case 11:
        // 启动平滑窗口大小为11的核函数
        gauss11SmoothImage<<<gridSize, blockSize>>>(origiImageGPU, 
                                                    gaussSmImageGPU,
                                                    smLocatX, smLocatY,
                                                    smWidth, smHeight);
        // 核函数出错
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }
            
        break;
         
    default:
        // 启动平滑窗口大小为3的核函数
        gauss3SmoothImage<<<gridSize,blockSize>>>(origiImageGPU, 
                                                  gaussSmImageGPU,
                                                  smLocatX, smLocatY,
                                                  smWidth, smHeight);
        // 核函数出错
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }
            
        break;
    }

    return NO_ERROR;

}

// 带mask的高斯平滑函数
__host__ int GaussSmoothImage::gaussSmoothImage(Image* origiImage, 
                                                int smWidth, int smHeight, 
                                                int smLocatX, int smLocatY, 
                                                int smWindowSize,
                                                Image* gaussSmImage, 
                                                Image* maskImage, 
                                                unsigned char mask)
{
    // 局部变量，错误码。
    int errcode;  

    // 获取图像尺寸信息
    int  imgWidth = origiImage->width;
    int  imgHeight = origiImage->height;

    // 图像小于平滑范围
    if (imgWidth < smWidth || imgHeight < smHeight) 
        return -11; 
        
    // 平滑范围小于最大平滑窗口
    if (smWidth < 11 || smHeight < 11) 
        return -12;
        
    // 输入的平滑窗口大小不在可处理范围之内
    if (smWindowSize < 3 || smWindowSize > 11) 
        return -13;

    // 平滑计算所涉及data位置或范围不能超出原始图像的物理范围，
    // 故应根据smWindowSize作适当调整。
    int  marginOff = (smWindowSize + 1) >> 1;
    int  leftMargin = smLocatX - marginOff;
    int  rightMargin = imgWidth - smLocatX - smWidth - marginOff;

    int  topMargin = smLocatY - marginOff;
    int  bottomMargin = imgHeight - smLocatY - smHeight - marginOff;

    // 平滑时将发生左侧出界
    if (leftMargin < 0) {
        smLocatX -= leftMargin;
        smWidth += leftMargin;
    }

    // 平滑时将发生右侧出界
    if (rightMargin < 0) {
        smWidth += rightMargin;
    }

    // 平滑宽度小于1
    if (smWidth < 1) 
       return -14; 

    // 平滑时将发生上方出界
    if (topMargin < 0) {
        smLocatY -= topMargin;
        smHeight += topMargin;
    }

    // 平滑时将发生下方出界
    if (bottomMargin < 0) {
        smHeight += bottomMargin;
    }

    // 平滑高度小于1
    if (smHeight < 1)
        return -15; 

    // 将输入图像拷贝到 Device 内存中。
    errcode = ImageBasicOp::copyToCurrentDevice(origiImage);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    // 将输出图像拷贝到 Device 内存中。
    errcode = ImageBasicOp::copyToCurrentDevice(gaussSmImage);
    if (errcode != NO_ERROR) {
        return errcode;
    }
    
    // 将mask图像拷贝到 Device 内存中。
    errcode = ImageBasicOp::copyToCurrentDevice(maskImage);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    // 提取输入图像。
    ImageCuda origiImageGPU;
    errcode = ImageBasicOp::roiSubImage(origiImage, &origiImageGPU);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    // 提取输出图像。
    ImageCuda gaussSmImageGPU;
    errcode = ImageBasicOp::roiSubImage(gaussSmImage, &gaussSmImageGPU);
    if (errcode != NO_ERROR) {
        return errcode;
    }
    
    // 提取mask图像。
    ImageCuda maskImageGPU;
    errcode = ImageBasicOp::roiSubImage(maskImage, &maskImageGPU);
    if (errcode != NO_ERROR) {
        return errcode;
    }
    
    // 计算调用 Kernel 函数的线程块的尺寸和线程块的数量。
    dim3 gridSize,blockSize;
    
    blockSize.x = DEF_BLOCK_X;
    blockSize.y = DEF_BLOCK_Y;
    gridSize.x = (smWidth + blockSize.x - 1) / blockSize.x;
    gridSize.y = (smHeight + blockSize.y - 1) / blockSize.y;

    // 根据平滑窗口大小选择对应的核函数
    // 按照委托方要求，顺序为7、5、9、11、3
    switch (smWindowSize)  { 
    
    case 7:
        // 启动平滑窗口大小为7的核函数
        gauss7SmoothImage<<<gridSize, blockSize>>>(origiImageGPU,
                                                   gaussSmImageGPU, 
                                                   smLocatX, smLocatY,
                                                   smWidth, smHeight,
                                                   maskImageGPU, mask); 
        // 核函数出错
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }
        break;
         
    case 5:
        // 启动平滑窗口大小为5的核函数
        gauss5SmoothImage<<<gridSize, blockSize>>>(origiImageGPU,
                                                   gaussSmImageGPU,
                                                   smLocatX, smLocatY,
                                                   smWidth, smHeight,
                                                   maskImageGPU, mask);
        // 核函数出错
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }
        break;
         
    case 9:
       
    // 每个窗口纵向线程块数目
 
        gauss9SmoothImage<<<gridSize, blockSize>>>(origiImageGPU, 
                                                   gaussSmImageGPU,
                                                   smLocatX, smLocatY,
                                                   smWidth, smHeight,
                                                   maskImageGPU, mask);
            // 核函数出错
            if (hipGetLastError() != hipSuccess) {
                return CUDA_ERROR;
            }
        break;
         
    case 11:
        // 启动平滑窗口大小为11的核函数
        gauss11SmoothImage<<<gridSize, blockSize>>>(origiImageGPU,
                                                    gaussSmImageGPU,
                                                    smLocatX, smLocatY,
                                                    smWidth, smHeight,
                                                    maskImageGPU, mask);
            // 核函数出错
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }
        break;
         
    default:
        // 启动平滑窗口大小为3的核函数
        gauss3SmoothImage<<<gridSize, blockSize>>>(origiImageGPU, 
                                                   gaussSmImageGPU,
                                                   smLocatX, smLocatY,
                                                   smWidth, smHeight,
                                                   maskImageGPU, mask);
        // 核函数出错
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }
        break;
    }

    return NO_ERROR;

}



